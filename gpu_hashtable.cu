#include "hip/hip_runtime.h"
#include <iostream>
#include <limits.h>
#include <stdlib.h>
#include <ctime>
#include <sstream>
#include <string>

#include "gpu_hashtable.hpp"

__global__ void insert(GpuHashTable::hashCell *deviceHashTable,
						unsigned int *keys,
						unsigned int *values,
						unsigned int slotsElems,
						int numKeys) {

	int index, position, idx;
	unsigned int old;

	idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (idx >= numKeys)
		return;

	position = myHash(keys[idx], slotsElems);
	index = position;
	while (index < slotsElems) {
		old = atomicCAS(&deviceHashTable[index].key, (unsigned int) 0, keys[idx]);
		if (old == 0 || old == keys[idx]) {
			deviceHashTable[index].value = values[idx];
			return;
		}	
		index++;
	}
	index = 0;
	while (index < position) {
		old = atomicCAS(&deviceHashTable[index].key, (unsigned int) 0, keys[idx]);
		if (old == 0 || old == keys[idx]) {
			deviceHashTable[index].value = values[idx];
			return;
		}	
		index++;
	}
}

__global__ void reinsert(GpuHashTable::hashCell *newHashTable,
						GpuHashTable::hashCell *copyHashTable,
						unsigned int oldSize,
						unsigned int slotsElems) {
	int index, position, idx;
	unsigned int old;
	
	idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= oldSize)
		return;
	
	if (copyHashTable[idx].key == 0)
		return;

	position = myHash(copyHashTable[idx].key, slotsElems);
	index = position;
	while (index < slotsElems) {
		old = atomicCAS(&newHashTable[index].key, (unsigned int) 0, copyHashTable[idx].key);
		if (!old || old == copyHashTable[idx].key) {
			newHashTable[index].value = copyHashTable[idx].value;
			return;
		}	
		index++;
	}
	index = 0;
	while (index < position) {
		old = atomicCAS(&newHashTable[index].key, (unsigned int) 0, copyHashTable[idx].key);
		if (!old || old == copyHashTable[idx].key) {
			newHashTable[index].value = copyHashTable[idx].value;
			return;
		}	
		index++;
	}
}

__global__ void get(GpuHashTable::hashCell *deviceHashTable,
					unsigned int *keys,
					unsigned int *values,
					unsigned int slotsElems,
					int numKeys) {
	int index, position, idx;

	idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= numKeys)
		return;

	position = myHash(keys[idx], slotsElems);
	index = position;
	while (index < slotsElems) {
		if (deviceHashTable[index].key == keys[idx]) {
			values[idx] = deviceHashTable[index].value;
			return;
		}	
		index++;
	}
	index = 0;
	while (index < position) {
		if (deviceHashTable[index].key == keys[idx]) {
			values[idx] = deviceHashTable[index].value;
			return;
		}	
		index++;
	}
}

/* INIT HASH
 */
GpuHashTable::GpuHashTable(int size) {

	hipMalloc((void **) &hashTable, size * sizeof(hashCell));
	hipMemset(hashTable, 0, size * sizeof(hashCell));

	slotsElems = size;
	numElems = 0;
}

/* DESTROY HASH
 */
GpuHashTable::~GpuHashTable() {
	hipFree(hashTable);
	slotsElems = 0;
	numElems = 0;
}

/* RESHAPE HASH
 */
void GpuHashTable::reshape(int numBucketsReshape) {
	hashCell *copyHashTable;

	if (numElems) {
		int mingridsize, threadblocksize, gridsize;
		hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize, reinsert, 0, 0);
		
		hipMalloc(&copyHashTable, slotsElems * sizeof(hashCell));
		hipMemcpy(copyHashTable, hashTable, slotsElems * sizeof(hashCell), hipMemcpyDeviceToDevice);

		hipFree(hashTable);

		hipMalloc((void **) &hashTable, numBucketsReshape * sizeof(hashCell));
		hipMemset(hashTable, 0, numBucketsReshape * sizeof(hashCell));

		gridsize = ((unsigned int)slotsElems + threadblocksize - 1) / threadblocksize;
		reinsert<<<gridsize, threadblocksize>>> (hashTable, copyHashTable, slotsElems, numBucketsReshape);
		hipDeviceSynchronize();
		slotsElems = numBucketsReshape;

		hipFree(copyHashTable);
		return;
	}

	hipFree(hashTable);

	hipMalloc((void **) &hashTable, numBucketsReshape * sizeof(hashCell));
	hipMemset(hashTable, 0, numBucketsReshape * sizeof(hashCell));
	slotsElems = numBucketsReshape;
}

/* INSERT BATCH
 */
bool GpuHashTable::insertBatch(int *keys, int *values, int numKeys) {
	int mingridsize;
	int threadblocksize;
	
    hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize, insert, 0, 0);
	int gridsize = ((unsigned int)numKeys + threadblocksize - 1) / threadblocksize;

	unsigned int *deviceKeys, *deviceValues;

	hipMalloc(&deviceKeys, numKeys * sizeof(int));
	hipMemcpy(deviceKeys, keys, numKeys * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc(&deviceValues, numKeys * sizeof(int));
	hipMemcpy(deviceValues, values, numKeys * sizeof(int), hipMemcpyHostToDevice);
	
	if ((float)(numElems + numKeys) / slotsElems > 0.95f)
		reshape((numElems + numKeys) * 1.25f);

	insert<<<gridsize, threadblocksize>>> (hashTable, deviceKeys, deviceValues, slotsElems, numKeys);
	hipDeviceSynchronize();

	hipFree(deviceKeys);
	hipFree(deviceValues);

	numElems += numKeys;
	return true;
}

/* GET BATCH
 */
int* GpuHashTable::getBatch(int* keys, int numKeys) {
	unsigned int *deviceKeys, *deviceValues;
	int *hostValues;

	int mingridsize, threadblocksize, gridsize;
	hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize, get, 0, 0);

	hipMalloc(&deviceKeys, numKeys * sizeof(int));
	hipMemcpy(deviceKeys, keys, numKeys * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc(&deviceValues, numKeys * sizeof(int));
	hipMemset(deviceValues, 0, numKeys * sizeof(int));

	hostValues = (int *) malloc(numKeys * sizeof(int));

	gridsize = ((unsigned int)numKeys + threadblocksize - 1) / threadblocksize;
	get<<<gridsize, threadblocksize>>> (hashTable, deviceKeys, deviceValues, slotsElems, numKeys);
	hipDeviceSynchronize();
	hipMemcpy(hostValues, deviceValues, numKeys * sizeof(int), hipMemcpyDeviceToHost);	

	hipFree(deviceKeys);
	hipFree(deviceValues);
	return hostValues;
}

/* GET LOAD FACTOR
 * num elements / hash total slots elements
 */
float GpuHashTable::loadFactor() {
	float loadFactor = 0.f;

	loadFactor = (float) numElems / slotsElems;

	return loadFactor; // no larger than 1.0f = 100%
}

/*********************************************************/

#define HASH_INIT GpuHashTable GpuHashTable(1);
#define HASH_RESERVE(size) GpuHashTable.reshape(size);

#define HASH_BATCH_INSERT(keys, values, numKeys) GpuHashTable.insertBatch(keys, values, numKeys)
#define HASH_BATCH_GET(keys, numKeys) GpuHashTable.getBatch(keys, numKeys)

#define HASH_LOAD_FACTOR GpuHashTable.loadFactor()

#include "test_map.cpp"
